#include "hip/hip_runtime.h"
// LBM Code for 2 - D, diffusion problems, D2Q4
// Adapted from the book Lattice Boltzmann Method - Fundamentals
// and Engineering Applications with Computer Codes by A. Mohamad
// Output file can be opened by the free software ParaView

// Eric Tada, April 24th, 2019

#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include <chrono>
#include "string.h"
#include "hip/hip_runtime.h"

#define m 100 //m is the number of lattice nodes (y)
#define n 100 //n is the number of lattice nodes (x)


// Collision kernel, done in parallel
__global__ void collision(float *f1, float *f2, float *f3, float *f4, float *rho, float *omega) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * (n+1);

	float feq = 0.25*rho[tid];
	f1[tid] = omega[0]*feq + (1.0 - omega[0])*f1[tid];
	f2[tid] = omega[0]*feq + (1.0 - omega[0])*f2[tid];
	f3[tid] = omega[0]*feq + (1.0 - omega[0])*f3[tid];
	f4[tid] = omega[0]*feq + (1.0 - omega[0])*f4[tid];
}

// Streaming kernel for f1 and f2, done in series for each row
__global__ void streaming12(float *f1, float *f2) {
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	for (int i = 1; i <= n; i++) {
		f1[j*(n + 1) + n - i] = f1[j*(n + 1) + n - i - 1];
		f2[j*(n+1) + i - 1] = f2[j*(n + 1) + i];
	}
}

// Streaming kernel for f3 and f4, done in series for each column
__global__ void streaming34(float *f3, float *f4) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	for (int j = 1; j <= m; j++) {
		f3[(m-j)*(n + 1) + i] = f3[(m - j - 1)*(n + 1) + i];
		f4[(j - 1)*(n + 1) + i] = f4[(j)*(n + 1) + i];
	}
}

// Kernel to apply boundary conditions (1)
__global__ void bound1(float *f1, float *f2, float *f3, float *f4) {
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j == 0 || j == n) {
	}
	else {
		f1[j*(n + 1)] = 0.5 - f2[j*(n + 1)];
		f3[j*(n + 1)] = 0.5 - f4[j*(n + 1)];
		f1[j*(n + 1) + n] = 0.0;
		f2[j*(n + 1) + n] = 0.0;
		f3[j*(n + 1) + n] = 0.0;
		f4[j*(n + 1) + n] = 0.0;
	}
}

// Kernel to apply boundary conditions (2)
__global__ void bound2(float *f1, float *f2, float *f3, float *f4) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i == 0 || i == n) {
	}
	else {
		f1[m*(n + 1) + i] = 0.0;
		f2[m*(n + 1) + i] = 0.0;
		f3[m*(n + 1) + i] = 0.0;
		f4[m*(n + 1) + i] = 0.0;
		f1[i] = f1[n + 1 + i];
		f2[i] = f2[n + 1 + i];
		f3[i] = f3[n + 1 + i];
		f4[i] = f4[n + 1 + i];
	}
}

// Kernel to update rho value
__global__ void update(float *f1, float *f2, float *f3, float *f4, float *rho) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * (n + 1);

	rho[tid] = f1[tid] + f2[tid] + f3[tid] + f4[tid];
}

// CPU function to output results
void print(float x[n + 1], float y[m + 1], float rho[(n + 1)*(m + 1)], int step) {
	char str[20];
	sprintf(str, "step%06d.vtk", step);

	FILE *res;
	res = fopen(str, "w");
	int i, j;

	fprintf(res, "# vtk DataFile Version 3.0\r\nvtk output\r\nASCII\r\nDATASET RECTILINEAR_GRID\r\nDIMENSIONS %d %d 1\r\n\r\n", n + 1, m + 1);
	fprintf(res, "X_COORDINATES %d  float\r\n", n + 1);
	for (i = 0; i <= n; i++) { fprintf(res, "%f ", x[i]); }
	fprintf(res, "\r\nY_COORDINATES %d  float\r\n", m + 1);
	for (j = 0; j <= m; j++) { fprintf(res, "%f ", y[j]); }
	fprintf(res, "\r\nZ_COORDINATES 1 float\r\n0\r\n\r\n");
	fprintf(res, "POINT_DATA %d\r\n", (n + 1)*(m + 1));
	fprintf(res, "FIELD FieldData 1\r\nv 1 %d float\r\n", (n + 1)*(m + 1));
	for (j = 0; j <= n; j++) {
		for (i = 0; i <= m; i++) {
			fprintf(res, "%f ", rho[j*(n+1) + i]);
		}
		fprintf(res, "\r\n");
	}
	fclose(res);
}

int main() {
	float f1[(n + 1)*(m + 1)], f2[(n + 1)*(m + 1)], f3[(n + 1)*(m + 1)], f4[(n + 1)*(m + 1)];
	float rho[(n + 1)*(m + 1)], x[n + 1], y[m + 1];
	int i, j;

	float dx = 1.0;
	float dy = dx;
	float dt = 1.0;

	x[0] = 0.0;
	y[0] = 0.0;
	for (i = 1; i <= n; i++) {
		x[i] = x[i - 1] + dx;
	}
	for (j = 1; j <= m; j++) {
		y[j] = y[j - 1] + dy;
	}
	float csq = dx*dx / (dt*dt);
	float alpha = 0.25;
	float omega[1];
	omega[0] = 1.0 / (2.*alpha / (dt*csq) + 0.5);
	float mstep = 4000;
	for (j = 0; j <= m; j++) {
		for (i = 0; i <= n; i++) {
			rho[j*(n+1) + i] = 0.0; //initial values of the dependent variable
		}
	}
	for (j = 0; j <= m; j++) {
		for (i = 0; i <= n; i++) {
			f1[j*(n + 1) + i] = 0.25*rho[j*(n + 1) + i];
			f2[j*(n + 1) + i] = 0.25*rho[j*(n + 1) + i];
			f3[j*(n + 1) + i] = 0.25*rho[j*(n + 1) + i];
			f4[j*(n + 1) + i] = 0.25*rho[j*(n + 1) + i];
		}
	}

	print(x, y, rho, 0);

	// Create GPU variables
	float *d_f1, float *d_f2, float *d_f3, float *d_f4, float *d_rho, float *d_omega;

	// Allocate memory to GPU
	hipMalloc((void**)&d_f1, ((n+1)*(m+1)) * sizeof(float));
	hipMalloc((void**)&d_f2, ((n + 1)*(m + 1)) * sizeof(float));
	hipMalloc((void**)&d_f3, ((n + 1)*(m + 1)) * sizeof(float));
	hipMalloc((void**)&d_f4, ((n + 1)*(m + 1)) * sizeof(float));
	hipMalloc((void**)&d_rho, ((n + 1)*(m + 1)) * sizeof(float));
	hipMalloc((void**)&d_omega, (1 * sizeof(float)));

	dim3 blocksij(n+1, m+1, 1), threads(1, 1, 1);
	dim3 blocksi(n + 1, 1, 1);
	dim3 blocksj(1, m + 1, 1);

	// Copy from host to device
	hipMemcpy(d_f1, f1, (n+1)*(m+1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_f2, f2, (n + 1)*(m + 1) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_f3, f3, (n + 1)*(m + 1) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_f4, f4, (n + 1)*(m + 1) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_rho, rho, (n + 1)*(m + 1) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_omega, omega, sizeof(float), hipMemcpyHostToDevice);

	// Record start time
	auto start = std::chrono::high_resolution_clock::now();

	for (int kk = 1; kk <= mstep; kk++) {

		//collision
		collision << < blocksij, threads >> > (d_f1, d_f2, d_f3, d_f4, d_rho, d_omega);

		//streaming
		streaming12 << <blocksj, threads >> > (d_f1, d_f2);
		streaming34 << <blocksi, threads >> > (d_f3, d_f4);

		//boundary conditions
		bound1 << <blocksj, threads >> > (d_f1, d_f2, d_f3, d_f4);
		bound2 << <blocksi, threads >> > (d_f1, d_f2, d_f3, d_f4);

		//update rho
		update << < blocksij, threads >> > (d_f1, d_f2, d_f3, d_f4, d_rho);

		//output result
		if (kk % 20 == 0) {
			hipMemcpy(rho, d_rho, (n + 1)*(m + 1) * sizeof(float), hipMemcpyDeviceToHost);
			print(x, y, rho, kk);
		}
	}

	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = finish - start;
	printf("Elapsed time for LBM: %f s\n", elapsed.count());

	getchar();
}